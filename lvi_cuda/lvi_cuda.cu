#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2014 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include "lvi_cuda.h"

#include <cmath>

#include <stdio.h>

// This is not C++0x, unfortunately.
#define nullptr NULL

// This is determined by hardware, so what is below is a 'safe' guess. If this is
// off, the program might return 'nan' or 'inf'.
#define FLT_MAX 1e+35

__global__ void lvi_bellman_update(unsigned int n, unsigned int z, const unsigned int *Pj,
		unsigned int m, const bool *A, const float *T, const float *R, float gamma,
		const float *Vi, float *ViPrime, unsigned int *pi)
{
	// The current state as a function of the blocks and threads.
	int s;

	// The intermediate Q(s, a) value.
	float Qsa;

	// The 1-d index version of the 3-d arrays in the innermost loop.
	int k;

	// Compute the index of the state. Return if it is beyond the state.
	s = blockIdx.x * blockDim.x + threadIdx.x;
	if (s >= z) {
		return;
	}

	// Nvidia GPUs follow IEEE floating point standards, so this should be safe.
	ViPrime[Pj[s]] = -FLT_MAX;

	// Compute max_{a in A} Q(s, a).
	for (int a = 0; a < m; a++) {
		// Skip this action if it is locked.
		if (!A[s * m + a]) {
			continue;
		}

		// Compute Q(s, a) for this action.
		Qsa = 0.0f;
		for (int sp = 0; sp < n; sp++) {
			k = Pj[s] * m * n + a * n + sp;
			Qsa += T[k] * (R[k] + gamma * Vi[sp]);
		}

		if (a == 0 || Qsa > ViPrime[Pj[s]]) {
			ViPrime[Pj[s]] = Qsa;
			pi[s] = a;
		}
	}
}

int lvi_cuda(unsigned int n, unsigned int z, unsigned int *Pj, unsigned int m, const bool *A,
		const float *T, const float *R, float Rmin, float Rmax, float gamma, float epsilon,
		float *Vi, unsigned int *pi, unsigned int numBlocks, unsigned int numThreads)
{
	// The device pointers for the MDP: A, T, and R.
	bool *d_A;
	float *d_T;
	float *d_R;

	// The host and device pointers for the value functions: V and VPrime.
	float *d_Vi;
	float *d_ViPrime;

	// The partition of states as an array of state indices.
	unsigned int *d_Pj;

	// The device pointer for the final policy: pi.
	unsigned int *d_pi;

	// First, ensure data is valid.
	if (n == 0 || z == 0 || Pj == nullptr || m == 0 || A == nullptr || T == nullptr || R == nullptr ||
			gamma < 0.0f || gamma >= 1.0f || pi == nullptr) {
		return -1;
	}

	// Also ensure that there are enough blocks and threads to run the program.
	if (numBlocks * numThreads < z) {
		return -2;
	}

	// Next, determine how many iterations it will have to run. Then, multiply that by 10.
	int iterations = 100; // (int)std::ceil(std::log(2.0 * (Rmax - Rmin) / (epsilon * (1.0 - gamma)) / std::log(1.0 / gamma)));

	// Allocate the device-side memory.
	if (hipMalloc(&d_A, z * m * sizeof(bool)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the restricted actions.");
		return -3;
	}
	if (hipMalloc(&d_T, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the state transitions.");
		return -3;
	}
	if (hipMalloc(&d_R, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the rewards.");
		return -3;
	}

	if (hipMalloc(&d_Vi, n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the value function.");
		return -3;
	}
	if (hipMalloc(&d_ViPrime, n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the value function (prime).");
		return -3;
	}

	if (hipMalloc(&d_Pj, z * sizeof(unsigned int)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the partition array.");
		return -3;
	}

	if (hipMalloc(&d_pi, z * sizeof(unsigned int)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the policy (pi).");
		return -3;
	}

//	// Assume that V and pi are initialized *properly* (either 0, or, with MPI, perhaps
//	// with previous V values).
//	for (int s = 0; s < n; s++) {
//		V[s] = 0.0f;
//		pi[s] = 0;
//	}

	// Copy the data from host to device.
	if (hipMemcpy(d_A, A, z * m * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the restricted actions.");
		return -3;
	}
	if (hipMemcpy(d_T, T, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the state transitions.");
		return -3;
	}
	if (hipMemcpy(d_R, R, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the rewards.");
		return -3;
	}

	if (hipMemcpy(d_Vi, Vi, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the value function.");
		return -3;
	}
	if (hipMemcpy(d_ViPrime, Vi, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the value function (prime).");
		return -3;
	}

	if (hipMemcpy(d_Pj, Pj, z * sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the partition array.");
		return -3;
	}

	if (hipMemcpy(d_pi, pi, z * sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the policy (pi).");
		return -3;
	}

	// Execute value iteration for these number of iterations. For each iteration, however,
	// we will run the state updates in parallel.
	printf("Total Number of Iterations: %i\n", iterations);
	for (int i = 0; i < iterations; i++) {
//		printf("Iteration %d / %d\n", i, iterations);
//		printf("Blocks: %d\nThreads: %d\nGamma: %f\nn: %d\nm: %d\n", numBlocks, numThreads, gamma, n, m);

		if (i % 2 == 0) {
			lvi_bellman_update<<< numBlocks, numThreads >>>(n, z, d_Pj, m, d_A, d_T, d_R, gamma, d_Vi, d_ViPrime, d_pi);
		} else {
			lvi_bellman_update<<< numBlocks, numThreads >>>(n, z, d_Pj, m, d_A, d_T, d_R, gamma, d_ViPrime, d_Vi, d_pi);
		}
	}

	// Copy the final result, both V and pi, from device to host.
	if (iterations % 2 == 1) {
		if (hipMemcpy(Vi, d_Vi, n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lvi_cuda]: %s",
					"Failed to copy memory from device to host for the value function.");
			return -3;
		}
	} else {
		if (hipMemcpy(Vi, d_ViPrime, n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lvi_cuda]: %s",
					"Failed to copy memory from device to host for the value function (prime).");
			return -3;
		}
	}
	if (hipMemcpy(pi, d_pi, z * sizeof(unsigned int), hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from device to host for the policy (pi).");
		return -3;
	}

//	for (int s = 0; s < n; s++) {
//		printf("V[%d] =   %f\t", s, V[s]);
//		if (s % 8 == 7) {
//			printf("\n");
//		}
//	}

	// Free the device-side memory.
	hipFree(d_A);
	hipFree(d_T);
	hipFree(d_R);

	hipFree(d_Vi);
	hipFree(d_ViPrime);

	hipFree(d_Pj);

	hipFree(d_pi);

	return 0;
}
