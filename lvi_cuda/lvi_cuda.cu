#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2014 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include "lvi_cuda.h"

#include <cmath>

#include <stdio.h>

// This is not C++0x, unfortunately.
#define nullptr NULL

// This is determined by hardware, so what is below is a 'safe' guess. If this is
// off, the program might return 'nan' or 'inf'.
#define FLT_MAX 1e+35

__global__ void lvi_bellman_update(unsigned int n, unsigned int z, unsigned int m,
		const bool *A, const unsigned int *Pj, const float *T, const float *Ri, unsigned int *pi,
		float gamma, const float *Vi,
		float *ViPrime)
{
	// The current state as a function of the blocks and threads.
	int s;

	// The intermediate Q(s, a) value.
	float Qsa;

	// The 1-d index version of the 3-d arrays in the innermost loop.
	int k;

	// Compute the index of the state. Return if it is beyond the state.
	s = blockIdx.x * blockDim.x + threadIdx.x;
	if (s >= z) {
		return;
	}

	// Nvidia GPUs follow IEEE floating point standards, so this should be safe.
	ViPrime[Pj[s]] = -FLT_MAX;

	// Compute max_{a in A} Q(s, a).
	for (int a = 0; a < m; a++) {
		// Skip this action if it is locked.
		if (!A[s * m + a]) {
			continue;
		}

		// Compute Q(s, a) for this action.
		Qsa = 0.0f;
		for (int sp = 0; sp < n; sp++) {
			k = Pj[s] * m * n + a * n + sp;
			Qsa += T[k] * (Ri[k] + gamma * Vi[sp]);
		}

		if (a == 0 || Qsa > ViPrime[Pj[s]]) {
			ViPrime[Pj[s]] = Qsa;
			pi[s] = a;
		}
	}
}

int lvi_initialize_state_transitions(unsigned int n, unsigned int m, const float *T, float *&d_T)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || T == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_T, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the state transitions.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_T, T, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the state transitions.");
		return -3;
	}

	return 0;
}

int lvi_initialize_rewards(unsigned int n, unsigned int m, const float *R, float *&d_R)
{
	// Ensure the data is valid.
	if (n == 0 || m == 0 || R == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_R, n * m * n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the rewards.");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_R, R, n * m * n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the rewards.");
		return -3;
	}

	return 0;
}

int lvi_initialize_partition(unsigned int z,
		const unsigned int *Pj, const unsigned int *pi,
		unsigned int *&d_Pj, unsigned int *&d_pi)
{
	// Ensure the data is valid.
	if (z == 0 || Pj == nullptr || pi == nullptr) {
		return -1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_Pj, z * sizeof(unsigned int)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the partition array.");
		return -3;
	}

	if (hipMalloc(&d_pi, z * sizeof(unsigned int)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the policy (pi).");
		return -3;
	}

	// Copy the data from the host to the device.
	if (hipMemcpy(d_Pj, Pj, z * sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the partition array.");
		return -3;
	}

	if (hipMemcpy(d_pi, pi, z * sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the policy (pi).");
		return -3;
	}

	return 0;
}

int lvi_get_policy(unsigned int z, const unsigned int *d_pi, unsigned int *pi)
{
	if (hipMemcpy(pi, d_pi, z * sizeof(unsigned int), hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from device to host for the policy (pi).");
		return -3;
	}

	return 0;
}

int lvi_uninitialize(float *&d_T,
		unsigned int k, float **&d_R,
		unsigned int ell, unsigned int **&d_P, unsigned int **&d_pi)
{
	hipFree(d_T);

	for (int i = 0; i < k ; i++) {
		hipFree(d_R[i]);
	}

	for (int j = 0; j < ell; j++) {
		hipFree(d_P[j]);
		hipFree(d_pi[j]);
	}

	return 0;
}

int lvi_cuda(unsigned int n, unsigned int z, unsigned int m, const bool *A,
		const float *d_T, const float *d_Ri, const unsigned int *d_Pj, unsigned int *d_pi,
		float Rmin, float Rmax, float gamma, float epsilon,
		unsigned int numBlocks, unsigned int numThreads,
		float *Vi)
{
	// The device pointers for the MDP: A, T, and R.
	bool *d_A;

	// The host and device pointers for the value functions: V and VPrime.
	float *d_Vi;
	float *d_ViPrime;

	// First, ensure data is valid.
	if (n == 0 || z == 0 || m == 0 || A == nullptr ||
			d_Pj == nullptr || d_T == nullptr || d_Ri == nullptr || d_pi == nullptr ||
			gamma < 0.0f || gamma >= 1.0f) {
		return -1;
	}

	// Also ensure that there are enough blocks and threads to run the program.
	if (numBlocks * numThreads < z) {
		return -2;
	}

	// Next, determine how many iterations it will have to run. Then, multiply that by 10.
	int iterations = max(10, (int)std::ceil(std::log(2.0 * (Rmax - Rmin) / (epsilon * (1.0 - gamma)) / std::log(1.0 / gamma))));

	// Allocate the device-side memory.
	if (hipMalloc(&d_A, z * m * sizeof(bool)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the restricted actions.");
		return -3;
	}

	if (hipMalloc(&d_Vi, n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the value function.");
		return -3;
	}
	if (hipMalloc(&d_ViPrime, n * sizeof(float)) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to allocate device-side memory for the value function (prime).");
		return -3;
	}

//	// Assume that V and pi are initialized *properly* (either 0, or, with MPI, perhaps
//	// with previous V values).
//	for (int s = 0; s < n; s++) {
//		V[s] = 0.0f;
//		pi[s] = 0;
//	}

	// Copy the data from host to device.
	if (hipMemcpy(d_A, A, z * m * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the restricted actions.");
		return -3;
	}

	if (hipMemcpy(d_Vi, Vi, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the value function.");
		return -3;
	}
	if (hipMemcpy(d_ViPrime, Vi, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error[lvi_cuda]: %s",
				"Failed to copy memory from host to device for the value function (prime).");
		return -3;
	}

	// Execute value iteration for these number of iterations. For each iteration, however,
	// we will run the state updates in parallel.
//	printf("Total Number of Iterations: %i\n", iterations);
	for (int i = 0; i < iterations; i++) {
//		printf("Iteration %d / %d\n", i, iterations);
//		printf("Blocks: %d\nThreads: %d\nGamma: %f\nn: %d\nm: %d\n", numBlocks, numThreads, gamma, n, m);

		if (i % 2 == 0) {
			lvi_bellman_update<<< numBlocks, numThreads >>>(n, z, m, d_A, d_Pj, d_T, d_Ri, d_pi, gamma, d_Vi, d_ViPrime);
		} else {
			lvi_bellman_update<<< numBlocks, numThreads >>>(n, z, m, d_A, d_Pj, d_T, d_Ri, d_pi, gamma, d_ViPrime, d_Vi);
		}
	}

	// Copy the final result, both V and pi, from device to host.
	if (iterations % 2 == 1) {
		if (hipMemcpy(Vi, d_Vi, n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lvi_cuda]: %s",
					"Failed to copy memory from device to host for the value function.");
			return -3;
		}
	} else {
		if (hipMemcpy(Vi, d_ViPrime, n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
			fprintf(stderr, "Error[lvi_cuda]: %s",
					"Failed to copy memory from device to host for the value function (prime).");
			return -3;
		}
	}

//	for (int s = 0; s < n; s++) {
//		printf("V[%d] =   %f\t", s, V[s]);
//		if (s % 8 == 7) {
//			printf("\n");
//		}
//	}

	// Free the device-side memory.
	hipFree(d_A);

	hipFree(d_Vi);
	hipFree(d_ViPrime);

	return 0;
}
